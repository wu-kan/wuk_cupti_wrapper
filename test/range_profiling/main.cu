#include "hip/hip_runtime.h"
#include "cupti_wrapper.h"
#include "helper_cupti.h"
#include <hip/hip_runtime.h>

template <typename T> __global__ void stupid_kernel(T *x, size_t n) {
  for (size_t i = blockDim.x * (size_t)blockIdx.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    x[i] += 1;
  }
}

template <typename T> class StupidWordload {
private:
  hipDeviceptr_t x;
  size_t n;
  hipStream_t s;

public:
  StupidWordload(size_t num = 1 << 20) : n(num) {
    DRIVER_API_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    DRIVER_API_CALL(hipMallocAsync(&x, sizeof(T) * n, s));
  }
  void sync() { DRIVER_API_CALL(hipStreamSynchronize(s)); }
  void reset() {
    DRIVER_API_CALL(hipMemsetD8Async(x, 0, sizeof(T) * n, s));
    sync();
  }
  void run_async() {
    stupid_kernel<T><<<(n + 255) / 256, 256, 0, s>>>((T *)x, n);
  }
  ~StupidWordload() {
    DRIVER_API_CALL(hipFreeAsync(x, s));
    sync();
    DRIVER_API_CALL(hipStreamDestroy(s));
  }
};

int main(int argc, char *argv[]) {
  hipDevice_t cuDevice;
  hipCtx_t cuContext;
  DRIVER_API_CALL(hipInit(0));
  DRIVER_API_CALL(hipDeviceGet(&cuDevice, 0));
  DRIVER_API_CALL(hipDevicePrimaryCtxRetain(&cuContext, cuDevice));
  DRIVER_API_CALL(hipCtxPushCurrent(cuContext));

  // Set up the workload
  do {
    StupidWordload<int> workload0;
    StupidWordload<float> workload1;
    auto reset = [&] {
      workload0.reset();
      workload1.reset();
    };
    auto run = [&] {
      workload0.run_async();
      workload1.run_async();
      workload0.sync();
      workload1.sync();
    };
    std::vector<std::string> metricNames = {"sm__cycles_elapsed.sum",
                                            "sm__cycles_active.sum"};
    do {
      wuk::CuProfiler p(metricNames);
      p.ProfileKernels("Workload", reset, run);
      std::string res = wuk::CuProfiler::res_to_json(p.MetricValues());
      std::fprintf(stdout, "%s", res.c_str());
    } while (0);
  } while (0);

  DRIVER_API_CALL(hipCtxPopCurrent(&cuContext));
  DRIVER_API_CALL(hipDevicePrimaryCtxRelease(cuDevice));
  return 0;
}
