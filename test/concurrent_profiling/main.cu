#include "hip/hip_runtime.h"
#include <cstddef>
#include <hip/hip_runtime.h>
#include <string>
#include <wuk/cupti_wrapper.hh>

template <typename T> __global__ void stupid_kernel(T *x, size_t n) {
  for (size_t i = blockDim.x * (size_t)blockIdx.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    x[i] += 1;
  }
}

#define DRIVER_API_CALL(x)                                                     \
  do {                                                                         \
    hipError_t result = x;                                                       \
    if (result != hipSuccess) {                                              \
      const char *msg;                                                         \
      hipDrvGetErrorName(result, &msg);                                            \
      std::fprintf(stderr,                                                     \
                   "{\"File\": "                                               \
                   "\"%s\", \"Line\": %d, \"Error\": \"%s\",\"Code\": %d, "    \
                   "\"Msg\": \"%s\"}\n",                                       \
                   __FILE__, __LINE__, #x, (int)result, msg);                  \
      std::exit(1);                                                            \
    }                                                                          \
  } while (0)

template <typename T> class StupidWordload {
private:
  hipDeviceptr_t x;
  size_t n;
  hipStream_t s;

public:
  StupidWordload(size_t num = 1 << 20) : n(num) {
    DRIVER_API_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    DRIVER_API_CALL(hipMallocAsync(&x, sizeof(T) * n, s));
  }
  void sync() { DRIVER_API_CALL(hipStreamSynchronize(s)); }
  void reset() {
    DRIVER_API_CALL(hipMemsetD8Async(x, 0, sizeof(T) * n, s));
    sync();
  }
  void run_async() {
    stupid_kernel<T><<<(n + 255) / 256, 256, 0, s>>>((T *)x, n);
  }
  ~StupidWordload() {
    DRIVER_API_CALL(hipFreeAsync(x, s));
    sync();
    DRIVER_API_CALL(hipStreamDestroy(s));
  }
};

int main() {
  hipDevice_t device;
  hipCtx_t ctx;
  DRIVER_API_CALL(hipInit(0));
  DRIVER_API_CALL(hipDeviceGet(&device, 0));
  DRIVER_API_CALL(hipDevicePrimaryCtxRetain(&ctx, device));
  DRIVER_API_CALL(hipCtxPushCurrent(ctx));
  do {
    StupidWordload<int> workload0;
    StupidWordload<float> workload1;
    auto reset = [&] {
      workload0.reset();
      workload1.reset();
    };
    auto run = [&] {
      workload0.run_async();
      workload1.run_async();
      workload0.sync();
      workload1.sync();
    };
    do {
      // https://docs.nvidia.com/cupti/main/main.html#metrics-mapping-table
      std::vector<std::string> metricNames{"sm__cycles_elapsed.sum",
                                           "sm__cycles_active.sum"};
      wuk::CuProfiler p(metricNames);
      p.ProfileKernels("range_name", reset, run);
      std::string res = wuk::CuProfiler::res_to_json(p.MetricValues());
      std::fprintf(stdout, "%s", res.c_str());
    } while (0);
  } while (0);
  DRIVER_API_CALL(hipCtxPopCurrent(&ctx));
  DRIVER_API_CALL(hipDevicePrimaryCtxRelease(device));
}