#include "hip/hip_runtime.h"
#include <cstddef>
#include <hip/hip_runtime.h>
#include <string>
#include <wuk/cupti_wrapper.hh>

template <typename T> __global__ void stupid_kernel(T *x, size_t n) {
  for (size_t i = blockDim.x * (size_t)blockIdx.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    x[i] += 1;
  }
}

#define CUDA_SAFE_CALL(x)                                                      \
  do {                                                                         \
    hipError_t result = x;                                                       \
    if (result != hipSuccess) {                                              \
      const char *msg;                                                         \
      hipDrvGetErrorName(result, &msg);                                            \
      std::fprintf(stderr,                                                     \
                   "{\"File\": "                                               \
                   "\"%s\", \"Line\": %d, \"Error\": \"%s\",\"Code\": %d, "    \
                   "\"Msg\": \"%s\"}\n",                                       \
                   __FILE__, __LINE__, #x, (int)result, msg);                  \
      std::exit(1);                                                            \
    }                                                                          \
  } while (0)

template <typename T> class StupidWordload {
private:
  hipDeviceptr_t x;
  size_t n;
  hipStream_t s;

public:
  StupidWordload(size_t num = 1 << 20) : n(num) {
    CUDA_SAFE_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    CUDA_SAFE_CALL(hipMallocAsync(&x, sizeof(T) * n, s));
  }
  void sync() { CUDA_SAFE_CALL(hipStreamSynchronize(s)); }
  void reset() {
    CUDA_SAFE_CALL(hipMemsetD8Async(x, 0, sizeof(T) * n, s));
    sync();
  }
  void run_async() {
    stupid_kernel<T><<<(n + 255) / 256, 256, 0, s>>>((T *)x, n);
  }
  ~StupidWordload() {
    CUDA_SAFE_CALL(hipFreeAsync(x, s));
    sync();
    CUDA_SAFE_CALL(hipStreamDestroy(s));
  }
};

int main() {
  hipDevice_t device;
  hipCtx_t ctx;
  CUDA_SAFE_CALL(hipInit(0));
  CUDA_SAFE_CALL(hipDeviceGet(&device, 0));
  CUDA_SAFE_CALL(hipDevicePrimaryCtxRetain(&ctx, device));
  CUDA_SAFE_CALL(hipCtxPushCurrent(ctx));
  do {
    StupidWordload<int> kernel0;
    StupidWordload<float> kernel1;
    auto reset = [&] {
      kernel0.reset();
      kernel1.reset();
    };
    auto run = [&] {
      kernel0.run_async();
      kernel1.run_async();
      kernel0.sync();
      kernel1.sync();
    };
    wuk::CuProfiler::init();
    do {
      // https://docs.nvidia.com/cupti/main/main.html#metrics-mapping-table
      std::vector<std::string> metricNames{"sm__cycles_elapsed.sum",
                                           "sm__cycles_active.sum"};
#if 0
      wuk::CuProfiler::ProfilingConfig cfg;
      cfg.maxRangeNameLength = 16; // the max length of "range_name"
      wuk::CuProfiler p(metricNames, cfg);
#else
      wuk::CuProfiler p(metricNames);
#endif
      p.ProfileKernels("range_name", reset, run);
      std::string res =
          wuk::CuProfiler::res_to_json(p.MetricValues(metricNames));
      std::fprintf(stdout, "%s", res.c_str());
    } while (0);
    wuk::CuProfiler::deinit();
  } while (0);
  CUDA_SAFE_CALL(hipCtxPopCurrent(&ctx));
  CUDA_SAFE_CALL(hipDevicePrimaryCtxRelease(device));
}