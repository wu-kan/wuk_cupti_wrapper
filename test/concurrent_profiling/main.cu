#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <wuk/cupti_wrapper.hh>

template <typename T> __global__ void stupid_kernel(T *x, size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    x[i] = 0;
  }
}

#define CUDA_SAFE_CALL(x)                                                      \
  do {                                                                         \
    hipError_t result = x;                                                       \
    if (result != hipSuccess) {                                              \
      const char *msg;                                                         \
      hipDrvGetErrorName(result, &msg);                                            \
      std::fprintf(stderr,                                                     \
                   "{\"File\": "                                               \
                   "\"%s\", \"Line\": %d, \"Error\": \"%s\",\"Code\": %d, "    \
                   "\"Msg\": \"%s\"}\n",                                       \
                   __FILE__, __LINE__, #x, (int)result, msg);                  \
      std::exit(1);                                                            \
    }                                                                          \
  } while (0)

template <typename T> struct StupidTester {
  T *x;
  size_t n;
  hipStream_t s;
  StupidTester(size_t num = 1 << 20) : n(num) {
    CUDA_SAFE_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    CUDA_SAFE_CALL(hipMalloc((hipDeviceptr_t *)&x, n * sizeof(T)));
  }
  void reset() { CUDA_SAFE_CALL(hipStreamSynchronize(s)); }
  void run() {
    stupid_kernel<T><<<(n + 255) / 256, 256, 0, s>>>(x, n);
    CUDA_SAFE_CALL(hipStreamSynchronize(s));
  }
  ~StupidTester() {
    CUDA_SAFE_CALL(hipFree((hipDeviceptr_t)x));
    CUDA_SAFE_CALL(hipStreamDestroy(s));
  }
};

int main() {
  hipDevice_t device;
  hipCtx_t context;
  CUDA_SAFE_CALL(hipInit(0));
  CUDA_SAFE_CALL(hipDeviceGet(&device, 0));
  CUDA_SAFE_CALL(hipCtxCreate(&context, 0, device));

  // https://docs.nvidia.com/cupti/main/main.html#metrics-mapping-table
  std::vector<std::string> metricNames{"sm__cycles_elapsed.sum",
                                       "sm__cycles_active.sum"};
  wuk::Profiler::init();
  wuk::Profiler p(metricNames);
  StupidTester<int> kernel0;
  StupidTester<float> kernel1;
  auto reset = [&] {
    kernel0.reset();
    kernel1.reset();
  };
  auto run = [&] {
    kernel0.run();
    kernel1.run();
  };
  p.ProfileKernels("RangeName", reset, run);
  auto res = p.MetricValuesToJSON(metricNames);
  std::fprintf(stdout, "%s", res.c_str());
  wuk::Profiler::deinit();
  // CUDA_SAFE_CALL(cuCtxDestory(context));
}